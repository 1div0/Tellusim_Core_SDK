#include "hip/hip_runtime.h"
// Copyright (C) 2018-2025, Tellusim Technologies Inc. All rights reserved
// https://tellusim.com/

#include <common/common.h>
#include <common/sample_controls.h>
#include <platform/TellusimDevice.h>
#include <platform/TellusimContext.h>
#include <platform/TellusimPipeline.h>
#include <platform/TellusimCommand.h>

/*
 */
__global__ void kernel(uint32_t size, float time, hipSurfaceObject_t surface) {
	
	uint32_t global_x = blockDim.x * blockIdx.x + threadIdx.x;
	uint32_t global_y = blockDim.y * blockIdx.y + threadIdx.y;
	
	float k = time * 2.0f;
	
	float s = sin(time * 0.5f);
	float c = cos(time * 0.5f);
	
	float2 t = make_float2((float)global_x / (float)size - 0.5f, (float)global_y / (float)size - 0.5f);
	
	t = make_float2((s * t.x + c * t.y) * 32.0f - 16.0f, (c * t.x - s * t.y) * 32.0f - 16.0f);
	
	float v = sin(t.x + k) + sin(t.y + k) + sin(t.x + t.y + k) + sin(sqrt(t.x * t.x + t.y * t.y) + k * 3.0f) + k * 2.0f;
	
	float r = (cos(v + 0.00f) * 0.5f + 0.5f) * 255.0f;
	float g = (cos(v + 1.57f) * 0.5f + 0.5f) * 255.0f;
	float b = (cos(v + 3.14f) * 0.5f + 0.5f) * 255.0f;
	uchar4 color = make_uchar4((uint8_t)r, (uint8_t)g, (uint8_t)b, 255);
	
	surf2Dwrite(color, surface, global_x * sizeof(uchar4), global_y);
}

/*
 */
int32_t main(int32_t argc, char **argv) {
	
	using namespace Tellusim;
	
	DECLARE_WINDOW
	
	// create window
	String title = String::format("%s Tellusim::CudaTexture", window.getPlatformName());
	if(!window.create(title) || !window.setHidden(false)) return 1;
	
	// create device
	Device device(window);
	if(!device) return 1;
	
	// create Cuda context for our device
	CUContext cu_context = CUContext(Context(PlatformCU, device.getFeatures().pciBusID));
	if(!cu_context || !cu_context.create()) {
		TS_LOG(Error, "main(): can't create Cuda context\n");
		return 1;
	}
	
	// create Cuda device
	Device cu_device(cu_context);
	if(!cu_device) return 1;
	
	// set Cuda device
	if(hipSetDevice(cu_context.getDevice()) != hipSuccess) return 1;
	
	// Cuda info
	int32_t driver_version = 0;
	int32_t runtime_version = 0;
	if(hipDriverGetVersion(&driver_version) != hipSuccess) return 1;
	if(hipRuntimeGetVersion(&runtime_version) != hipSuccess) return 1;
	TS_LOGF(Message, "Driver: %u\n", driver_version);
	TS_LOGF(Message, "Runtime: %u\n", runtime_version);
	
	// create pipeline
	Pipeline pipeline = device.createPipeline();
	pipeline.setSamplerMask(0, Shader::MaskFragment);
	pipeline.setTextureMask(0, Shader::MaskFragment);
	pipeline.setColorFormat(window.getColorFormat());
	pipeline.setDepthFormat(window.getDepthFormat());
	if(!pipeline.loadShaderGLSL(Shader::TypeVertex, "main.shader", "VERTEX_SHADER=1")) return 1;
	if(!pipeline.loadShaderGLSL(Shader::TypeFragment, "main.shader", "FRAGMENT_SHADER=1")) return 1;
	if(!pipeline.create()) return 1;
	
	// create sampler
	Sampler sampler = device.createSampler(Sampler::FilterLinear, Sampler::WrapModeClamp);
	if(!sampler) return 1;
	
	// create texture
	constexpr uint32_t texture_size = 1024;
	Texture texture = device.createTexture2D(FormatRGBAu8n, texture_size, Texture::FlagSurface | Texture::FlagInterop);
	if(!texture) return 1;
	
	// create Cuda texture
	CUTexture cu_texture = CUTexture(cu_device.createTexture(texture));
	if(!cu_texture) return 1;
	
	// create Cuda surface desc
	hipResourceDesc surface_desc = {};
	surface_desc.resType = hipResourceTypeArray;
	surface_desc.res.array.array = (hipArray_t)cu_texture.getTextureLevel(0);
	
	// create Cuda surface
	hipSurfaceObject_t cu_surface = 0;
	hipError_t error = hipCreateSurfaceObject(&cu_surface, &surface_desc);
	if(error != hipSuccess) return 1;
	
	// create target
	Target target = device.createTarget(window);
	
	// main loop
	DECLARE_GLOBAL
	window.run([&]() -> bool {
		DECLARE_COMMON
		
		// suppress warnings
		simulate = simulate;
		pause = pause;
		
		Window::update();
		
		if(!window.render()) return false;
		
		// window title
		if(fps > 0.0f) window.setTitle(String::format("%s %.1f FPS", title.get(), fps));
		
		// dispatch Cuda kernel
		{
			// dispatch Cuda kernel
			uint32_t group_size = 8;
			uint32_t num_groups = udiv(texture_size, group_size);
			hipStream_t stream = (hipStream_t)cu_context.getStream();
			kernel<<<dim3(num_groups, num_groups), dim3(group_size, group_size), 0, stream>>>(texture_size, time, cu_surface);
			
			// check Cuda error
			hipError_t error = hipGetLastError();
			if(error != hipSuccess) TS_LOGF(Error, "main(): %s\n", hipGetErrorString(error));
			
			// synchronize stream
			hipStreamSynchronize(stream);
		}
		
		// flush texture
		device.flushTexture(texture);
		
		// window target
		target.begin();
		{
			// create command list
			Command command = device.createCommand(target);
			
			// draw texture
			command.setPipeline(pipeline);
			command.setSampler(0, sampler);
			command.setTexture(0, texture);
			command.drawArrays(3);
		}
		target.end();
		
		if(!window.present()) return false;
		
		if(!device.check()) return false;
		
		return true;
	});
	
	// finish context
	window.finish();
	
	return 0;
}
